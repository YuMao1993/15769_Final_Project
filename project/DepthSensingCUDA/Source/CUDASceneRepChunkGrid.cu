#include "hip/hip_runtime.h"

#include <cutil_inline.h>
#include <cutil_math.h>

#include "cuda_SimpleMatrixUtil.h"


#include "VoxelUtilHashSDF.h"
#include "DepthCameraUtil.h"

#define T_PER_BLOCK 8

struct SDFBlockDesc {
	int3 pos;
	int ptr;
};

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Streaming from GPU to CPU: copies only selected blocks/hashEntries to the CPU if outside of the frustum //
/////////////////////////////////////////////////////////////////////////////////////////////////////////////


//-------------------------------------------------------
// Pass 1: Find all SDFBlocks that have to be transfered
//-------------------------------------------------------

__global__ void integrateFromGlobalHashPass1Kernel(VoxelHashData voxelHashData, uint start, float radius, float3 cameraPosition, uint* d_outputCounter, SDFBlockDesc* d_output) 
{
	const HashParams& hashParams = c_hashParams;
	unsigned int hashEntryIdx = blockIdx.x*blockDim.x + threadIdx.x + start;
	const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (hashEntryIdx < hashParams.m_hashNumBuckets*HASH_BUCKET_SIZE) {
		HashEntry& entry = voxelHashData.d_hash[hashEntryIdx];
		float3 posWorld = voxelHashData.SDFBlockToWorld(entry.pos);
		float d = length(posWorld - cameraPosition);

		if (entry.ptr != FREE_ENTRY && d >= radius) {
			// Write
			SDFBlockDesc d;
			d.pos = entry.pos;
			d.ptr = entry.ptr;

			#ifndef HANDLE_COLLISIONS
				uint addr = atomicAdd(&d_outputCounter[0], 1);
				d_output[addr] = d;
				voxelHashData.appendHeap(entry.ptr/linBlockSize);
				voxelHashData.resetHashEntry(bucketID);
			#endif
			#ifdef HANDLE_COLLISIONS
				//if there is an offset or hash doesn't belong to the bucket (linked list)
				if (entry.offset != 0 || voxelHashData.computeHashPos(entry.pos) != hashEntryIdx / HASH_BUCKET_SIZE) {					
					if (voxelHashData.deleteHashEntry(entry.pos)) {
						voxelHashData.appendHeap(d.ptr / linBlockSize);
						uint addr = atomicAdd(&d_outputCounter[0], 1);
						d_output[addr] = d;
					}
				} else {
					uint addr = atomicAdd(&d_outputCounter[0], 1);
					d_output[addr] = d;
					voxelHashData.appendHeap(d.ptr / linBlockSize);
					voxelHashData.resetHashEntry(entry);
				}
			#endif
		}
	}
}

extern "C" void integrateFromGlobalHashPass1CUDA(const HashParams& hashParams, const VoxelHashData& voxelHashData, uint threadsPerPart, uint start, float radius, const float3& cameraPosition, uint* d_outputCounter, SDFBlockDesc* d_output)
{
	const dim3 gridSize((threadsPerPart + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	if (threadsPerPart > 0) {
		// each thread will check on one hash entry
		integrateFromGlobalHashPass1Kernel<<<gridSize, blockSize>>>(voxelHashData, start, radius, cameraPosition, d_outputCounter, d_output);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}


//-------------------------------------------------------
// Pass 2: Copy SDFBlocks to output buffer
//-------------------------------------------------------


__global__ void integrateFromGlobalHashPass2Kernel(VoxelHashData voxelHashData, const SDFBlockDesc* d_SDFBlockDescs, Voxel* d_output, unsigned int nSDFBlocks)
{
	const uint idxBlock = blockIdx.x;

	if (idxBlock < nSDFBlocks) {

		const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
		const uint idxInBlock = threadIdx.x;
		const SDFBlockDesc& desc = d_SDFBlockDescs[idxBlock];

		// Copy SDF block to CPU
		d_output[idxBlock*linBlockSize + idxInBlock] = voxelHashData.d_SDFBlocks[desc.ptr + idxInBlock];

		// Reset SDF Block
		voxelHashData.resetVoxel(desc.ptr + idxInBlock);
	}
}

extern "C" void integrateFromGlobalHashPass2CUDA(const HashParams& hashParams, const VoxelHashData& voxelHashData, uint threadsPerPart, const SDFBlockDesc* d_SDFBlockDescs, Voxel* d_output, unsigned int nSDFBlocks)
{
	const uint threadsPerBlock = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
	const dim3 gridSize(threadsPerPart, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	if (threadsPerPart > 0) {
		integrateFromGlobalHashPass2Kernel<<<gridSize, blockSize>>>(voxelHashData, d_SDFBlockDescs, d_output, nSDFBlocks);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}



///////////////////////////////////////////////////////////////////////
// Streaming from CPU to GPU: copies an entire chunk back to the GPU //
///////////////////////////////////////////////////////////////////////



//-------------------------------------------------------
// Pass 1: Allocate memory on GPU heap and insert hash entry.
//-------------------------------------------------------

__global__ void  chunkToGlobalHashPass1Kernel(VoxelHashData voxelHashData, uint numSDFBlockDescs, uint heapCountPrev, const SDFBlockDesc* d_SDFBlockDescs, const Voxel* d_SDFBlocks)
{
	const unsigned int sdfBlockIdx = blockIdx.x*blockDim.x + threadIdx.x;
	const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	if (sdfBlockIdx < numSDFBlockDescs)	{
		
		uint ptr = voxelHashData.d_heap[heapCountPrev - sdfBlockIdx] * linBlockSize;	//mass alloc

		HashEntry entry;
		entry.pos = d_SDFBlockDescs[sdfBlockIdx].pos;
		entry.offset = 0;
		entry.ptr = ptr;

		voxelHashData.insertHashEntry(entry);
	}
}

extern "C" void chunkToGlobalHashPass1CUDA(const HashParams& hashParams, const VoxelHashData& voxelHashData, uint numSDFBlockDescs, uint heapCountPrev, const SDFBlockDesc* d_SDFBlockDescs, const Voxel* d_SDFBlocks)
{
	const dim3 gridSize((numSDFBlockDescs + (T_PER_BLOCK*T_PER_BLOCK) - 1)/(T_PER_BLOCK*T_PER_BLOCK), 1);
	const dim3 blockSize((T_PER_BLOCK*T_PER_BLOCK), 1);

	if (numSDFBlockDescs > 0) {
		chunkToGlobalHashPass1Kernel<<<gridSize, blockSize>>>(voxelHashData, numSDFBlockDescs, heapCountPrev, d_SDFBlockDescs, d_SDFBlocks);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}

//-------------------------------------------------------
// Pass 2: Copy input to SDFBlocks
//-------------------------------------------------------

__global__ void chunkToGlobalHashPass2Kernel(VoxelHashData voxelHashData, uint heapCountPrev, const SDFBlockDesc* d_SDFBlockDescs, const Voxel* d_SDFBlocks)
{
	const uint blockID = blockIdx.x;
	const uint linBlockSize = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
		 
	uint ptr = voxelHashData.d_heap[heapCountPrev-blockID]*linBlockSize;

	int freeBlockIdx = voxelHashData.d_heap[heapCountPrev - blockID];
	//if (freeBlockIdx >= 262144) {
	//	printf("blocks idx: %d, heap idx: %d\n", freeBlockIdx, heapCountPrev - blockID);
	//}
	voxelHashData.d_SDFBlocks[ptr + threadIdx.x] = d_SDFBlocks[blockIdx.x*blockDim.x + threadIdx.x];
}


extern "C" void chunkToGlobalHashPass2CUDA(const HashParams& hashParams, const VoxelHashData& voxelHashData, uint numSDFBlockDescs, uint heapCountPrev, const SDFBlockDesc* d_SDFBlockDescs, const Voxel* d_SDFBlocks)
{
	const uint threadsPerBlock = SDF_BLOCK_SIZE * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;
	const dim3 gridSize(numSDFBlockDescs, 1);
	const dim3 blockSize(threadsPerBlock, 1);

	if (numSDFBlockDescs > 0) {
		// each thread is responsible for one voxel
		chunkToGlobalHashPass2Kernel<<<gridSize, blockSize>>>(voxelHashData, heapCountPrev, d_SDFBlockDescs, d_SDFBlocks);
	}

#ifdef _DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif
}